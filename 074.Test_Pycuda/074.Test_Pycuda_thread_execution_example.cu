#include "hip/hip_runtime.h"
#include <iostream>

#include <stdio.h>
__global__ void myfirstkernel(void) {
	//blockIdx.x gives the block number of current kernel
	printf("Hello!!!I'm thread in block: %d\n", blockIdx.x);
}

int main(void) {
	//A kernel call with 16 blocks and 1 thread per block
	myfirstkernel << <16,1>> >();
	//Function used for waiting for all kernels to finish
	hipDeviceSynchronize();
	printf("All threads are finished!\n");
	return 0;
}
