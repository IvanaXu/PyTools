#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
//Kernel function to add two variables, parameters are passed by reference
__global__ void gpuAdd(int *d_a, int *d_b, int *d_c) {
	*d_c = *d_a + *d_b;
}

int main(void) {
	//Defining host variables
	int h_a,h_b, h_c;
	//Defining Device Pointers
	int *d_a,*d_b,*d_c;
	//Initializing host variables
	h_a = 1;
	h_b = 4;
	//Allocating memory for Device Pointers
	hipMalloc((void**)&d_a, sizeof(int));
	hipMalloc((void**)&d_b, sizeof(int));
	hipMalloc((void**)&d_c, sizeof(int));
	//Coping value of host variables in device memory
	hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice);
	//Calling kernel with one thread and one block with parameters passed by reference
	gpuAdd << <1, 1 >> > (d_a, d_b, d_c);
	//Coping result from device memory to host
	hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);
	printf("Passing Parameter by Reference Output: %d + %d = %d\n", h_a, h_b, h_c);
	//Free up memory 
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}
