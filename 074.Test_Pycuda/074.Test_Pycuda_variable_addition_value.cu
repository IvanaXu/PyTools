#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

//Definition of kernel function to add two variables
__global__ void gpuAdd(int d_a, int d_b, int *d_c) {
	*d_c = d_a + d_b;
}

//main function
int main(void) {
	//Defining host variable to store answer
	int h_c;
	//Defining device pointer
	int *d_c;
	//Allocating memory for device pointer
	hipMalloc((void**)&d_c, sizeof(int));
	//Kernel call by passing 1 and 4 as inputs and storing answer in d_c
	//<< <1,1> >> means 1 block is executed with 1 thread per block
	gpuAdd << <1, 1 >> > (1, 4, d_c);
	//Copy result from device memory to host memory
	hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);
	printf("1 + 4 = %d\n", h_c);
	//Free up memory
	hipFree(d_c);
	return 0;
}