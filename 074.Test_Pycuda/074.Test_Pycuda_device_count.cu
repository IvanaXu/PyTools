#include <memory>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>

// Main Program 

int main(void)
{
	int device_Count = 0;
	hipGetDeviceCount(&device_Count);
	// This function returns count of number of CUDA enable devices and 0 if there are no CUDA capable devices.
	if (device_Count == 0)
	{
		printf("There are no available device(s) that support CUDA\n");
	}
	else
	{
		printf("Detected %d CUDA Capable device(s)\n", device_Count);
	}

	
}
