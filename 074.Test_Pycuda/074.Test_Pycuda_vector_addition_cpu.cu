
#include <hip/hip_runtime.h>
#include "stdio.h"
#include<iostream>
//Defining Number of elements in Array
#define N	5
//Defining vector addition function for CPU
void cpuAdd(int *h_a, int *h_b, int *h_c) {
	int tid = 0;	
	while (tid < N)
	{
		h_c[tid] = h_a[tid] + h_b[tid];
		tid += 1;
	}
}

int main(void) {
	int h_a[N], h_b[N], h_c[N];
		//Initializing two arrays for addition
	for (int i = 0; i < N; i++) {
		h_a[i] = 2 * i*i;
		h_b[i] = i;
	}
	//Calling CPU function for vector addition
	cpuAdd (h_a, h_b, h_c);
	//Printing Answer
	printf("Vector addition on CPU\n");
	for (int i = 0; i < N; i++) {
		printf("The sum of %d element is %d + %d = %d\n", i, h_a[i], h_b[i], h_c[i]);
	}
	return 0;
}
